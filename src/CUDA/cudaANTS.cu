#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#include <stdlib.h>

//=============== declarations  ===============

extern "C" bool cuda_run(const int Method,  //0 - Axial 2D; 1 - XY
                         const int blockSizeXY,
                         const int iterations,
                         const float scale,
                         const float scaleReductionFactor,
                         const int mlORchi2,
                         const bool ignoreLowSigPMs,
                         const float ignoreThresholdLow,
                         const float ignoreThresholdHigh,
                         const bool ignoreFarPMs,
                         const float ignoreDistance,
                         const float comp_r0,
                         const float comp_a,
                         const float comp_b,
                         const float comp_lam2,
                         const float* const PMx,
                         const float* const PMy,
                         const int numPMs,
                         const float* lrfdata,
                         const int lrfFloatsPerPM,
                         const int p1,
                         const int p2,
                         const int lrfFloatsAxialPerPM,  //only for Composite!
                         const float* EventsData,
                         const int numEvents,
                         float *RecX,
                         float *RecY,
                         float *RecEnergy,
                         float *Chi2,
                         float *Probability,
                         float *ElapsedTime);

__global__ void kernelRadial2D(const bool mlORchi2,
                               const bool ignoreLowSignalPMs,
                               const float ignoreThresholdLow,
                               const float ignoreThresholdHigh,
                               const bool ignoreFarPMs,
                               const float ignoreDistance2,
                               const int iterations,
                               float scale,
                               const float scaleReductionFactor,               
                               const float* pmx,
                               const float* pmy,
                               int numPMs,
                               int lrfSizePerPM,
                               const float * const d_lrfData,
                               const float* const d_eventsData,
                               int numEvents,
                               float* d_recX,
                               float* d_recY,
                               float* d_recEnergy,
                               float* d_chi2);

__global__ void kernelRadial2Dcomp(const bool mlORchi2,
                               const bool ignoreLowSignalPMs,
                               const float ignoreThresholdLow,
                               const float ignoreThresholdHigh,
                               const bool ignoreFarPMs,
                               const float ignoreDistance2,
                               const int iterations,
                               float scale,
                               const float scaleReductionFactor,
                               const float comp_r0,
                               const float comp_a,
                               const float comp_b,
                               const float comp_lam2,
                               const float* pmx,
                               const float* pmy,
                               int numPMs,
                               int lrfSizePerPM,
                               const float * const d_lrfData,
                               const float* const d_eventsData,
                               int numEvents,
                               float* d_recX,
                               float* d_recY,
                               float* d_recEnergy,
                               float* d_chi2);

__global__ void kernelComposite(const bool mlORchi2,
                                const bool ignoreLowSigPMs,
                                const float ignoreThresholdLow,
                                const float ignoreThresholdHigh,
                                const bool ignoreFarPMs,
                                const float ignoreDistance2,
                                const int iterations,
                                float scale,
                                const float scaleReductionFactor,
                                const float comp_r0,
                                const float comp_a,
                                const float comp_b,
                                const float comp_lam2,
                                const bool fCompressed,
                                const float* pmx,
                                const float* pmy,
                                int numPMs,
                                int lrfFloatsPerPM,
                                int lrfFloatsAxialPerPM,
                                int nintx,
                                int ninty,
                                const float* const d_lrfData,
                                const float* const d_eventsData,
                                int numEvents,
                                float* d_recX,
                                float* d_recY,
                                float* d_recEnergy,
                                float* d_chi2,
                                float* d_probability);

__global__ void kernelXY(const bool mlORchi2,
                               const bool ignoreLowSignalPMs,
                               const float ignoreThresholdLow,
                               const float ignoreThresholdHigh,
                         const bool ignoreFarPMs,
                         const float ignoreDistance2,
                         const float* pmx,
                         const float* pmy,
                               const int iterations,
                               float scale,
                               const float scaleReductionFactor,
                               int numPMs,
                               int lrfSizePerPM,
                               const int nintx,
                               const int ninty,
                               const float * const d_lrfData,
                               const float* const d_eventsData,
                               int numEvents,
                               float* d_recX,
                               float* d_recY,
                               float* d_recEnergy,
                               float* d_chi2,
                               float *d_probability);


//__constant__ float d_LRF[16384];  //constant memory - former was containing LRF data


//=============== implementation ===============

static char lastError[100];

//void checkCUDAError(const char* msg)
bool checkCUDAError(const char* msg)  //true if error found
{
   // Print a message if a CUDA error occurred
   hipError_t err = hipGetLastError();

   if (hipSuccess != err)
     {
      fprintf(stderr,"\n-->CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));      
      sprintf(lastError, "CUDA ERROR: %s: %s.\n", msg, hipGetErrorString(err));
      //exit(EXIT_FAILURE);
      return true;
     }
   else
     {
       sprintf(lastError, "%s", "");
     }
   return false;
}

extern "C" const char *getLastCUDAerror()
{
  return lastError;
}

extern "C" bool cuda_run(const int Method,  //0 - Axial 2D; 1 - XY; 3 - CompRad
                         const int blockSizeXY,
                         const int iterations,
                         const float scale,
                         const float scaleReductionFactor,
                         const int mlORchi2,
                         const bool ignoreLowSigPMs,
                         const float ignoreThresholdLow,
                         const float ignoreThresholdHigh,
                         const bool ignoreFarPMs,
                         const float ignoreDistance,
                         const float comp_r0,
                         const float comp_a,
                         const float comp_b,
                         const float comp_lam2,
                         const float* const PMx,
                         const float* const PMy,
                         const int numPMs,
                         const float* lrfdata,
                         const int lrfFloatsPerPM,
                         const int p1,
                         const int p2,
                         const int lrfFloatsAxialPerPM,
                         const float* EventsData,
                         const int numEvents,
                         float *RecX,
                         float *RecY,
                         float *RecEnergy,
                         float *Chi2,
                         float *Probability,
                         float *ElapsedTime)
{  
  //Allocate global memory on GPU for input/output event data
  int sizeEventsBuffer  = numEvents * (numPMs+2) * sizeof(float); //active PMs + XY of offset
  int sizeEvents = numEvents * sizeof(float);
  int sizePMsFloat = numPMs * sizeof(float);
  int sizeLRFsFloat = numPMs * lrfFloatsPerPM * sizeof(float);

  // Device data
  float* d_eventsData = 0;
  float* d_pmx = 0;
  float* d_pmy = 0;
  float* d_lrfData = 0;

  float* d_recX = 0;
  float* d_recY = 0;
  float* d_recEnergy = 0;
  float* d_chi2 = 0;
  float* d_probability = 0;

  fprintf(stderr,"cuda==> Device memory allocation\n");
    //input
  hipMalloc((void**) &d_eventsData, sizeEventsBuffer);
  hipMalloc((void**) &d_lrfData, sizeLRFsFloat);

  if (checkCUDAError("Memory alloc (eventsData bufer)")) return false;
  if (Method == 0 || Method == 3 || Method == 4)
    {
      hipMalloc((void**) &d_pmx, sizePMsFloat);
      if (checkCUDAError("Memory alloc (pmx)")) return false;
      hipMalloc((void**) &d_pmy, sizePMsFloat);
      if (checkCUDAError("Memory alloc (pmy)")) return false;
    }

    //output
  hipMalloc((void**) &d_recX, sizeEvents);
  if (checkCUDAError("Memory alloc (recX)")) return false;
  hipMalloc((void**) &d_recY, sizeEvents);
  if (checkCUDAError("Memory alloc (recY)")) return false;
  hipMalloc((void**) &d_recEnergy, sizeEvents);
  if (checkCUDAError("Memory alloc (recEnergy)")) return false;
  hipMalloc((void**) &d_chi2, sizeEvents);
  if (checkCUDAError("Memory alloc (chi2)")) return false;
  if (mlORchi2 == 0) //ML only
    {
      hipMalloc((void**) &d_probability, sizeEvents);
      if (checkCUDAError("Memory alloc (probability)")) return false;
    }


  // copy events to GPU global memory
  hipMemcpy(d_eventsData, EventsData, sizeEventsBuffer, hipMemcpyHostToDevice);
  if (checkCUDAError("Copy events data to GPU (events data)")) return false;
  if (Method == 0 || Method == 3 || Method == 4)
    {
      hipMemcpy(d_pmx, PMx, sizePMsFloat, hipMemcpyHostToDevice);
      if (checkCUDAError("Copy events data to GPU (PMx)")) return false;
      hipMemcpy(d_pmy, PMy, sizePMsFloat, hipMemcpyHostToDevice);
      if (checkCUDAError("Copy events data to GPU (PMy)")) return false;
    }

/*
  //setting up constant memory
  fprintf(stderr,"cuda==> LRF floats Per PM: %i )\n",lrfFloatsPerPM);
  int lrfdatasize = numPMs * lrfFloatsPerPM * sizeof(float);
  fprintf(stderr,"cuda==> Allocation of the constant memory (%i bytes)\n", lrfdatasize);
  hipMemcpyToSymbol(HIP_SYMBOL(d_LRF), lrfdata, lrfdatasize);
  if (checkCUDAError("Copy LRF data to constant memory")) return false;
*/
  hipMemcpy(d_lrfData, lrfdata, sizeLRFsFloat, hipMemcpyHostToDevice);
  if (checkCUDAError("Copy LRF data to GPU")) return false;

  dim3 threads(blockSizeXY,blockSizeXY,1);

  //calculating total ammount of needed shared memory
  int sizeSharedMem = 0;
  if (Method == 0 || Method == 3 || Method == 4) sizeSharedMem += numPMs * 2; //PMx and PMy
  sizeSharedMem += numPMs; //one-event data
  if (mlORchi2 == 0) sizeSharedMem += blockSizeXY*blockSizeXY; //probability - ML only
  sizeSharedMem += blockSizeXY*blockSizeXY; // X
  sizeSharedMem += blockSizeXY*blockSizeXY; //  Y
  sizeSharedMem += blockSizeXY*blockSizeXY; //  Energy
  sizeSharedMem += blockSizeXY*blockSizeXY; //  Chi2
  sizeSharedMem *= 4; // all data on shared use 32 bit words
  fprintf(stderr,"cuda==> Calculated ammount of shared memory usage: %i bytes\n", sizeSharedMem);

  int blocksX, blocksY;
  if (numEvents <  65536)
    {
      blocksX = numEvents;
      blocksY = 1;
    }
  else
    {
      blocksY = numEvents / 65535;
      if (numEvents % 65535 != 0) blocksY++;
      blocksX = numEvents / blocksY;
      if (numEvents % blocksY != 0) blocksX++;
    }
  fprintf(stderr,"cuda==> Invoking kernel with (%i, %i) grid and (%i, %i) threads...\n", blocksX, blocksY, blockSizeXY, blockSizeXY);
  dim3 grid(blocksX, blocksY, 1);

  //time control
  float elapsedTime;
  hipEvent_t start;
  hipEvent_t stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  //running kernel
  switch (Method)
    {
    case 0: // Axial
      kernelRadial2D <<< grid, threads, sizeSharedMem>>>(mlORchi2,
                                                         ignoreLowSigPMs,
                                                         ignoreThresholdLow,
                                                         ignoreThresholdHigh,
                                                         ignoreFarPMs,
                                                         ignoreDistance*ignoreDistance,
                                                         iterations,
                                                         scale,
                                                         scaleReductionFactor,                                             
                                                         d_pmx,
                                                         d_pmy,
                                                         numPMs,
                                                         lrfFloatsPerPM,
                                                         d_lrfData,
                                                         d_eventsData,
                                                         numEvents,
                                                         d_recX,
                                                         d_recY,
                                                         d_recEnergy,
                                                         d_chi2);

      break;
    case 3: //CompRad
      kernelRadial2Dcomp <<< grid, threads, sizeSharedMem>>>(mlORchi2,
                                                         ignoreLowSigPMs,
                                                         ignoreThresholdLow,
                                                         ignoreThresholdHigh,
                                                         ignoreFarPMs,
                                                         ignoreDistance*ignoreDistance,
                                                         iterations,
                                                         scale,
                                                         scaleReductionFactor,                                                         
                                                         comp_r0,
                                                         comp_a,
                                                         comp_b,
                                                         comp_lam2,
                                                         d_pmx,
                                                         d_pmy,
                                                         numPMs,
                                                         lrfFloatsPerPM,
                                                         d_lrfData,
                                                         d_eventsData,
                                                         numEvents,
                                                         d_recX,
                                                         d_recY,
                                                         d_recEnergy,
                                                         d_chi2);

      break;
    case 1:
      //kernelFreeform <<< grid, threads, sizeSharedMem>>>(mlORchi2,
            kernelXY <<< grid, threads, sizeSharedMem>>>(mlORchi2,
                                                         ignoreLowSigPMs,
                                                         ignoreThresholdLow,
                                                         ignoreThresholdHigh,
                                                       ignoreFarPMs,
                                                       ignoreDistance*ignoreDistance,
                                                       d_pmx,
                                                       d_pmy,
                                                         iterations,
                                                         scale,
                                                         scaleReductionFactor,
                                                         numPMs,
                                                         lrfFloatsPerPM,
                                                         p1, // nintx
                                                         p2, // ninty
                                                         d_lrfData,
                                                         d_eventsData,
                                                         numEvents,
                                                         d_recX,
                                                         d_recY,
                                                         d_recEnergy,
                                                         d_chi2,
                                                         d_probability);

      break;
    case 4:
      kernelComposite <<< grid, threads, sizeSharedMem>>>(mlORchi2,
                                                         ignoreLowSigPMs,
                                                         ignoreThresholdLow,
                                                         ignoreThresholdHigh,
                                                         ignoreFarPMs,
                                                         ignoreDistance*ignoreDistance,
                                                         iterations,
                                                         scale,
                                                         scaleReductionFactor,
                                                         comp_r0,
                                                         comp_a,
                                                         comp_b,
                                                         comp_lam2,
                                                         (comp_r0!=0 || comp_a!=0 || comp_b!=0 || comp_lam2!=0),
                                                         d_pmx,
                                                         d_pmy,
                                                         numPMs,
                                                         lrfFloatsPerPM,
                                                         lrfFloatsAxialPerPM,
                                                         p1, // nintx
                                                         p2, // ninty
                                                         d_lrfData,
                                                         d_eventsData,
                                                         numEvents,
                                                         d_recX,
                                                         d_recY,
                                                         d_recEnergy,
                                                         d_chi2,
                                                         d_probability);
      break;
    default:
      break;
    }

  hipDeviceSynchronize();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  *ElapsedTime = elapsedTime;

  if (checkCUDAError("Kernel invocations")) return false;
  fprintf(stderr,"cuda==> Kernel invocation successful\n");

  fprintf(stderr,"cuda==> kernel execution elapsed time: %f ms\n", elapsedTime);


  // copy results to CPU memory
  hipMemcpy(RecX, d_recX, sizeEvents, hipMemcpyDeviceToHost);
  if (checkCUDAError("Copy results to CPU (recX)")) return false;
  hipMemcpy(RecY, d_recY, sizeEvents, hipMemcpyDeviceToHost);
  if (checkCUDAError("Copy results to CPU (recY)")) return false;
  hipMemcpy(RecEnergy, d_recEnergy, sizeEvents, hipMemcpyDeviceToHost);
  if (checkCUDAError("Copy results to CPU (recEnergy)")) return false;
  hipMemcpy(Chi2, d_chi2, sizeEvents, hipMemcpyDeviceToHost);
  if (checkCUDAError("Copy results to CPU (chi2)")) return false;
  if (mlORchi2 == 0) //ML only
    {
      hipMemcpy(Probability, d_probability, sizeEvents, hipMemcpyDeviceToHost);
      if (checkCUDAError("Copy results to CPU (probability)")) return false;
    }

  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(d_eventsData);
  if (Method == 0 || Method == 3)
    {
      hipFree(d_pmx);
      hipFree(d_pmy);
    }
  hipFree(d_recX);
  hipFree(d_recY);
  hipFree(d_recEnergy);
  hipFree(d_chi2);
  if (mlORchi2 == 0) hipFree(d_probability); //ML only
  if (checkCUDAError("Cuda memory free")) return false;

  return true;
}

__global__ void kernelRadial2D(const bool mlORchi2,
                               const bool ignoreLowSignalPMs,
                               const float ignoreThresholdLow,
                               const float ignoreThresholdHigh,
                               const bool ignoreFarPMs,
                               const float ignoreDistance2,
                               const int iterations,
                               float scale,
                               const float scaleReductionFactor,                 
                               const float* pmx,
                               const float* pmy,
                               int numPMs,
                               int lrfSizePerPM, //intervals + 1
                               const float* const d_lrfData,
                               const float* const d_eventsData,
                               int numEvents,
                               float* d_recX,
                               float* d_recY,
                               float* d_recEnergy,
                               float* d_chi2)

{
  //setting up shared memory
  int activeThreads = blockDim.x * blockDim.y;

  extern __shared__ float shared[];  //these data are shared withing a thread block - that is for all points of the grid (one event!)
  float* PMx = (float*) &shared;
  float* PMy = (float*) &PMx[numPMs];
  float* signal = (float*) &PMy[numPMs];
  int* nodeIX = (int*) &signal[numPMs];
  int* nodeIY = (int*) &nodeIX[activeThreads];
  float* nodeChi2 = (float*) &nodeIY[activeThreads];
  float* nodeEnergy = (float*) &nodeChi2[activeThreads];
    //for ML only
  float* nodeProb = (float*) &nodeEnergy[activeThreads];

  __shared__ float Xoffset;
  __shared__ float Yoffset;
  __shared__ float Xoffset0; //will be kept unchanged, for dynamic passives by distance
  __shared__ float Yoffset0; //will be kept unchanged, for dynamic passives by distance

  int ievent = blockIdx.x + gridDim.x * blockIdx.y;      //cuda GRID is used to select an event - below word "grid" refers to the grid of XYs positions
  int threadID = threadIdx.x + threadIdx.y * blockDim.x; //block is used to scan XY (each node - one thread)
                                                         //threadIdx.x is related to offset in X from Xcenter of the grid
                                                         //threadIdx.y is related to offset in Y from Ycenter of the grid

  //to do!!! case when numPMs>numtreads in block
  if (threadID<numPMs)
    { //in this block _only_: threadID is PMs index
      PMx[threadID] = pmx[threadID];
      PMy[threadID] = pmy[threadID];

      signal[threadID] = d_eventsData[ievent*(numPMs+2) + threadID]; //buffer contains PM signals and XY offset
    }

  if (threadID == 0)
    { //0th tread sets the center of the grid
      Xoffset = d_eventsData[ievent*(numPMs+2) + numPMs];  //offsetX;
      Xoffset0 = Xoffset;
      Yoffset = d_eventsData[ievent*(numPMs+2) + numPMs+1];//offsetY;
      Yoffset0 = Yoffset;
    }

  __syncthreads(); //need to synchronize - all shared input data are ready

  int activePMs;

  //starting multigrid iterations
  for (int iter=0; iter<iterations; iter++)
   {
     float X = Xoffset + scale * (-0.5*(blockDim.x - 1)+ threadIdx.x); //coordinates of this node (Thread)
     float Y = Yoffset + scale * (-0.5*(blockDim.y - 1)+ threadIdx.y);

     if (ievent<numEvents)
       {
         activePMs = 0;
         //since we cannot save lrfs in a fast tmp storage, collecting all statistics in parralel
         //energy - SUM sig(i) / SUM LRF(i)
         float sumsig = 0;
         float sumLRF = 0;
         //probability for ML: SUM sig(i)*ln{LRF(i)*E} - LRF(i)*E = SUM   sig(i)*ln(LRF(i)) + sig(i)*ln(E) - LRF(i)*E
         float sumsigLnLRF = 0;
         //for chi2:  SUM   {sig(i) - LRF(i)*E)^2 / LRF(i)E  =
         //           SUM   sig2/lrf/E -2*sig +lrf*E
         float sumsig2overLRF = 0;

         bool isBadNode = false; //can be set to true if range to one of PMs larger than the defininition range of its LRF; or energy < 0
         for (int ipm=0; ipm<numPMs; ipm++)
           {
             float tsig = signal[ipm];
             if (ignoreLowSignalPMs)
                if (tsig < ignoreThresholdLow || tsig > ignoreThresholdHigh)
                 {
                  //signal < threshold, ignoring this PM
                  continue;
                 }
             if (ignoreFarPMs)
               {
                 float stX  = (PMx[ipm] - Xoffset0);
                 float stY  = (PMy[ipm] - Yoffset0);
                 float r2 = stX*stX + stY*stY;// r  = sqrtf(r);
                 //distance2 from this PM center to the start position of the search
                 if (r2 > ignoreDistance2)
                  {
                   //too far from this PM center, ignoring this PM
                   continue;
                  }
               }


             float dx = d_lrfData[lrfSizePerPM*ipm + lrfSizePerPM-1]; //LRFs are defined in the range from 0 to dx
             float offX  = (X - PMx[ipm]);
             float offY  = (Y - PMy[ipm]);
             float r = offX*offX + offY*offY;
             r  = sqrtf(r); //distance to the PM center
             if (r > dx)
               {
                 //outside! cannot use this thread (node) at all
                 isBadNode = true;
                 break;
               }

             activePMs++; //this one is active
             sumsig += tsig;             

             //LRF calculation
             if (r == dx) dx -= 1.0e-7;
             int nint = lrfSizePerPM-4; // lrfSizePerPM = number of intervals +3 + 1
             float xi = r/dx*nint;
             int ix = (int)xi;
             float xf = xi-ix;

             float a0, a1, a2;
             float p0, p1, p2, p3;

             a0 = d_lrfData[lrfSizePerPM*ipm + ix];
             a1 = d_lrfData[lrfSizePerPM*ipm + ix + 1];
             float c2 = d_lrfData[lrfSizePerPM*ipm + ix + 2];
             a2 = a0 + a1 + c2;  // c0 + c1 + c2
             a1 = a1 + a1 + a1;  // 3*c1
             p0 = a2 + a1;       // c0 + 4*c1 + c2
             p2 = a2 - a1;       // c0 - 2*c1 + c2
             p2 = p2 + p2 + p2;  // 3*c0 - 6*c1 + 3*c2
             p1 = c2 - a0;       // c2 - c0
             p1 = p1 + p1 + p1;  // 3*c2 - 3*c0
             p3 = a0 + a0 - a1 - p2 + d_lrfData[lrfSizePerPM*ipm + ix + 3]; // -c0 + 3*c1 - 3*c2 + c3

             float lrf = (p0 + xf*(p1 + xf*(p2 + xf*p3)))/6.;

             //have lrf now
             if (lrf <= 0)
               { //bad lrf
                 isBadNode = true;
                 break;
               }
             sumLRF += lrf;
             if (mlORchi2 == 0) sumsigLnLRF += tsig * __logf(lrf);
             sumsig2overLRF += tsig * tsig / lrf;

           } //end cycle by PMs

         //can calculate energy ("naive" approach)
         float energy = 1.0;
         if (sumLRF > 0 ) energy = sumsig/sumLRF;
         if (energy<1.0e-10) isBadNode = true;

         //can calculate probability and chi2
         //storing results in shared memory
         if (isBadNode)
           {
             if (mlORchi2 == 0) nodeProb[threadID] = -1.0e10;
             nodeChi2[threadID] = 1.0e10;
           }
         else
           {
             if (mlORchi2 == 0) nodeProb[threadID] = sumsigLnLRF  +  sumsig * __logf(energy)  -  sumLRF*energy;
             nodeChi2[threadID] = sumsig2overLRF/energy -2*sumsig + sumLRF*energy;
           }

         nodeIX[threadID] = threadIdx.x;
         nodeIY[threadID] = threadIdx.y;
         nodeEnergy[threadID] = energy;
       }
     else
       {
         //could happen for num events >65535
         //wrong event index - we dont care, these data are not sent to host
         //only important: these threads still can see all __syncthreads()
       }

     __syncthreads(); //to prepare node data

     //looking for the best location - compare nodes always in pairs - using shared memory
      int Nodes = activeThreads;    //activeThreads = blockDim.x * blockDim.y
      do
              {
                int idx = Nodes / 2;
                //---

                if (mlORchi2 == 0) //ML
                  {
                    if (threadID < idx && nodeProb[threadID] < nodeProb[threadID + idx])
                      {
                       nodeProb[threadID] = nodeProb[threadID + idx];
                       nodeIX[threadID] = nodeIX[threadID + idx];
                       nodeIY[threadID] = nodeIY[threadID + idx];
                       nodeEnergy[threadID] = nodeEnergy[threadID + idx];
                       nodeChi2[threadID] = nodeChi2[threadID + idx];
                      }

                    if (threadID == 0)
                     if (Nodes % 2 != 0)  //for the cases when Nodes is odd number
                      {
                        if (nodeProb[0] < nodeProb[Nodes-1]) //all other nodes are already covered
                          {
                           nodeProb[0] = nodeProb[Nodes-1];
                           nodeIX[0] = nodeIX[Nodes-1];
                           nodeIY[0] = nodeIY[Nodes-1];
                           nodeEnergy[0] = nodeEnergy[Nodes-1];
                           nodeChi2[0] = nodeChi2[Nodes-1];
                          }
                      }
                  }
                else
                  {
                    if (threadID < idx && nodeChi2[threadID] > nodeChi2[threadID + idx])
                      {
                       nodeIX[threadID] = nodeIX[threadID + idx];
                       nodeIY[threadID] = nodeIY[threadID + idx];
                       nodeEnergy[threadID] = nodeEnergy[threadID + idx];
                       nodeChi2[threadID] = nodeChi2[threadID + idx];
                      }

                    if (threadID == 0)
                     if (Nodes % 2 != 0)  //for the cases when Nodes is odd number
                      {
                        if (nodeChi2[0] > nodeChi2[Nodes-1]) //all other nodes are already covered
                          {
                           nodeIX[0] = nodeIX[Nodes-1];
                           nodeIY[0] = nodeIY[Nodes-1];
                           nodeEnergy[0] = nodeEnergy[Nodes-1];
                           nodeChi2[0] = nodeChi2[Nodes-1];
                          }
                      }
                  }


                //---
                Nodes = idx;
                __syncthreads();
              }
            while (Nodes>1);

     // thread index 0 contains info on to the best fit location with the current grid
     if (threadID == 0)
       {
         Xoffset +=  scale * (-0.5*(blockDim.x - 1)+ nodeIX[0]); //new gid center - for the next iteration
         Yoffset +=  scale * (-0.5*(blockDim.y - 1)+ nodeIY[0]);
       }
     __syncthreads();


     //preparing scale for the next iteration
     scale /= scaleReductionFactor;
   }//iteration end

  //reporting back reconstructed event data
  if (threadID == 0 && ievent<numEvents)
      {
        //reporting results
        d_recX[ievent] = Xoffset;
        d_recY[ievent] = Yoffset;
        d_recEnergy[ievent] = nodeEnergy[0];
        int df = activePMs - 4;  //-1 -2XY -1energy
        if (df < 1) df = 1;
        d_chi2[ievent] = nodeChi2[0] / df;
      }
}

//it is identical to the kernelRadial2 except handling radial compression
__global__ void kernelRadial2Dcomp(const bool mlORchi2,
                               const bool ignoreLowSignalPMs,
                               const float ignoreThresholdLow,
                               const float ignoreThresholdHigh,
                               const bool ignoreFarPMs,
                               const float ignoreDistance2,
                               const int iterations,
                               float scale,
                               const float scaleReductionFactor,
                               const float comp_r0,
                               const float comp_a,                               
                               const float comp_b,
                               const float comp_lam2,
                               const float* pmx,
                               const float* pmy,
                               int numPMs,
                               int lrfSizePerPM,                                   
                               const float* const d_lrfData,
                               const float* const d_eventsData,
                               int numEvents,
                               float* d_recX,
                               float* d_recY,
                               float* d_recEnergy,
                               float* d_chi2)
{
  //setting up shared memory
  int activeThreads = blockDim.x * blockDim.y;

  extern __shared__ float shared[];  //these data are shared withing a thread block - that is for all points of the grid (one event!)
  float* PMx = (float*) &shared;
  float* PMy = (float*) &PMx[numPMs];
  float* signal = (float*) &PMy[numPMs];
  int* nodeIX = (int*) &signal[numPMs];
  int* nodeIY = (int*) &nodeIX[activeThreads];
  float* nodeChi2 = (float*) &nodeIY[activeThreads];
  float* nodeEnergy = (float*) &nodeChi2[activeThreads];
    //for ML only
  float* nodeProb = (float*) &nodeEnergy[activeThreads];

  __shared__ float Xoffset;
  __shared__ float Yoffset;
  __shared__ float Xoffset0; //will be kept unchanged, for dynamic passives by distance
  __shared__ float Yoffset0; //will be kept unchanged, for dynamic passives by distance

  int ievent = blockIdx.x + gridDim.x * blockIdx.y;      //cuda GRID is used to select an event - below word "grid" refers to the grid of XYs positions
  int threadID = threadIdx.x + threadIdx.y * blockDim.x; //block is used to scan XY (each node - one thread)
                                                         //threadIdx.x is related to offset in X from Xcenter of the grid
                                                         //threadIdx.y is related to offset in Y from Ycenter of the grid

  //to do!!! case when numPMs>numtreads in block
  if (threadID<numPMs)
    { //in this block _only_: threadID is PMs index
      PMx[threadID] = pmx[threadID];
      PMy[threadID] = pmy[threadID];

      signal[threadID] = d_eventsData[ievent*(numPMs+2) + threadID]; //buffer contains PM signals and XY offset
    }

  if (threadID == 0)
    { //0th tread sets the center of the grid
      Xoffset = d_eventsData[ievent*(numPMs+2) + numPMs];  //offsetX;
      Xoffset0 = Xoffset;
      Yoffset = d_eventsData[ievent*(numPMs+2) + numPMs+1];//offsetY;
      Yoffset0 = Yoffset;
    }

  __syncthreads(); //need to synchronize - all shared input data are ready

  int activePMs;

  //starting multigrid iterations
  for (int iter=0; iter<iterations; iter++)
   {
     float X = Xoffset + scale * (-0.5*(blockDim.x - 1)+ threadIdx.x); //coordinates of this node (Thread)
     float Y = Yoffset + scale * (-0.5*(blockDim.y - 1)+ threadIdx.y);

     if (ievent<numEvents)
       {
         activePMs = 0;
         //since we cannot save lrfs in a fast tmp storage, collecting all statistics in parralel
         //energy - SUM sig(i) / SUM LRF(i)
         float sumsig = 0;
         float sumLRF = 0;
         //probability for ML: SUM sig(i)*ln{LRF(i)*E} - LRF(i)*E = SUM   sig(i)*ln(LRF(i)) + sig(i)*ln(E) - LRF(i)*E
         float sumsigLnLRF = 0;
         //for chi2:  SUM   {sig(i) - LRF(i)*E)^2 / LRF(i)E  =
         //           SUM   sig2/lrf/E -2*sig +lrf*E
         float sumsig2overLRF = 0;

         bool isBadNode = false; //can be set to true if range to one of PMs larger than the defininition range of its LRF; or energy < 0
         for (int ipm=0; ipm<numPMs; ipm++)
           {
             float tsig = signal[ipm];
             if (ignoreLowSignalPMs)
                if (tsig < ignoreThresholdLow || tsig > ignoreThresholdHigh)
                 {
                  //signal < threshold, ignoring this PM
                  continue;
                 }
             if (ignoreFarPMs)
               {
                 float stX  = (PMx[ipm] - Xoffset0);
                 float stY  = (PMy[ipm] - Yoffset0);
                 float r2 = stX*stX + stY*stY; // r  = sqrtf(r);
                 //distance2 from this PM center to the start position of the search
                 if (r2 > ignoreDistance2)
                  {
                   //too far from this PM center, ignoring this PM
                   continue;
                  }
               }

             float dx = d_lrfData[lrfSizePerPM*ipm + lrfSizePerPM-1]; //LRFs are defined in the range from 0 to dx
             float offX  = (X - PMx[ipm]);
             float offY  = (Y - PMy[ipm]);
             float r = offX*offX + offY*offY;
             r  = sqrtf(r); //distance to the PM center            
             if (r > dx)
               {
                 //outside! cannot use this thread (node) at all
                 isBadNode = true;
                 break;
               }

             activePMs++; //this one is active
             sumsig += tsig;

             //applying compression
             float delta_r = r - comp_r0;
             r =  comp_b + delta_r*comp_a - sqrtf(delta_r*delta_r + comp_lam2);
             if (r<0.0) r = 0.0;

             delta_r = dx - comp_r0;
             dx =  comp_b + delta_r*comp_a - sqrtf(delta_r*delta_r + comp_lam2);
           // if (dx<0.0) dx = 0.0;


             //LRF calculation
             if (r == dx) dx -= 1.0e-7;
             int nint = lrfSizePerPM-4; // lrfSizePerPM = number of intervals +3 + 1
             float xi = r/dx*nint;
             int ix = (int)xi;
             float xf = xi-ix;

             float a0, a1, a2;
             float p0, p1, p2, p3;

             a0 = d_lrfData[lrfSizePerPM*ipm + ix];
             a1 = d_lrfData[lrfSizePerPM*ipm + ix + 1];
             float c2 = d_lrfData[lrfSizePerPM*ipm + ix + 2];
             a2 = a0 + a1 + c2;  // c0 + c1 + c2
             a1 = a1 + a1 + a1;  // 3*c1
             p0 = a2 + a1;       // c0 + 4*c1 + c2
             p2 = a2 - a1;       // c0 - 2*c1 + c2
             p2 = p2 + p2 + p2;  // 3*c0 - 6*c1 + 3*c2
             p1 = c2 - a0;       // c2 - c0
             p1 = p1 + p1 + p1;  // 3*c2 - 3*c0
             p3 = a0 + a0 - a1 - p2 + d_lrfData[lrfSizePerPM*ipm + ix + 3]; // -c0 + 3*c1 - 3*c2 + c3

             float lrf = (p0 + xf*(p1 + xf*(p2 + xf*p3)))/6.;

             //have lrf now
             if (lrf <= 0)
               { //bad lrf
                 isBadNode = true;
                 break;
               }
             sumLRF += lrf;
             if (mlORchi2 == 0) sumsigLnLRF += tsig * __logf(lrf);
             sumsig2overLRF += tsig * tsig / lrf;

           } //end cycle by PMs

         //can calculate energy ("naive" approach)
         float energy = 1.0;
         if (sumLRF > 0 ) energy = sumsig/sumLRF;
         if (energy<1.0e-10) isBadNode = true;

         //can calculate probability and chi2
         //storing results in shared memory
         if (isBadNode)
           {
             if (mlORchi2 == 0) nodeProb[threadID] = -1.0e10;
             nodeChi2[threadID] = 1.0e10;
           }
         else
           {
             if (mlORchi2 == 0) nodeProb[threadID] = sumsigLnLRF  +  sumsig * __logf(energy)  -  sumLRF*energy;
             nodeChi2[threadID] = sumsig2overLRF/energy -2*sumsig + sumLRF*energy;
           }

         nodeIX[threadID] = threadIdx.x;
         nodeIY[threadID] = threadIdx.y;
         nodeEnergy[threadID] = energy;
       }
     else
       {
         //could happen for num events >65535
         //wrong event index - we dont care, these data are not sent to host
         //only important: these threads still can see all __syncthreads()
       }

     __syncthreads(); //to prepare node data

     //looking for the best location - compare nodes always in pairs - using shared memory
      int Nodes = activeThreads;    //activeThreads = blockDim.x * blockDim.y
      do
              {
                int idx = Nodes / 2;
                //---

                if (mlORchi2 == 0) //ML
                  {
                    if (threadID < idx && nodeProb[threadID] < nodeProb[threadID + idx])
                      {
                       nodeProb[threadID] = nodeProb[threadID + idx];
                       nodeIX[threadID] = nodeIX[threadID + idx];
                       nodeIY[threadID] = nodeIY[threadID + idx];
                       nodeEnergy[threadID] = nodeEnergy[threadID + idx];
                       nodeChi2[threadID] = nodeChi2[threadID + idx];
                      }

                    if (threadID == 0)
                     if (Nodes % 2 != 0)  //for the cases when Nodes is odd number
                      {
                        if (nodeProb[0] < nodeProb[Nodes-1]) //all other nodes are already covered
                          {
                           nodeProb[0] = nodeProb[Nodes-1];
                           nodeIX[0] = nodeIX[Nodes-1];
                           nodeIY[0] = nodeIY[Nodes-1];
                           nodeEnergy[0] = nodeEnergy[Nodes-1];
                           nodeChi2[0] = nodeChi2[Nodes-1];
                          }
                      }
                  }
                else
                  {
                    if (threadID < idx && nodeChi2[threadID] > nodeChi2[threadID + idx])
                      {
                       nodeIX[threadID] = nodeIX[threadID + idx];
                       nodeIY[threadID] = nodeIY[threadID + idx];
                       nodeEnergy[threadID] = nodeEnergy[threadID + idx];
                       nodeChi2[threadID] = nodeChi2[threadID + idx];
                      }

                    if (threadID == 0)
                     if (Nodes % 2 != 0)  //for the cases when Nodes is odd number
                      {
                        if (nodeChi2[0] > nodeChi2[Nodes-1]) //all other nodes are already covered
                          {
                           nodeIX[0] = nodeIX[Nodes-1];
                           nodeIY[0] = nodeIY[Nodes-1];
                           nodeEnergy[0] = nodeEnergy[Nodes-1];
                           nodeChi2[0] = nodeChi2[Nodes-1];
                          }
                      }
                  }


                //---
                Nodes = idx;
                __syncthreads();
              }
            while (Nodes>1);

     // thread index 0 contains info on to the best fit location with the current grid
     if (threadID == 0)
       {
         Xoffset +=  scale * (-0.5*(blockDim.x - 1)+ nodeIX[0]); //new gid center - for the next iteration
         Yoffset +=  scale * (-0.5*(blockDim.y - 1)+ nodeIY[0]);
       }
     __syncthreads();


     //preparing scale for the next iteration
     scale /= scaleReductionFactor;
   }//iteration end

  //reporting back reconstructed event data
  if (threadID == 0 && ievent<numEvents)
      {
        //reporting results
        d_recX[ievent] = Xoffset;
        d_recY[ievent] = Yoffset;
        d_recEnergy[ievent] = nodeEnergy[0];
        int df = activePMs -4;  //-1 -2XY -1energy
        if (df < 1) df = 1;
        d_chi2[ievent] = nodeChi2[0] / df;
      }
}

/*
__global__ void kernelFreeform(const bool mlORchi2,
                               const bool ignoreLowSignalPMs,
                               const float ignoreThreshold,
                               const int iterations,
                               float scale,
                               const float scaleReductionFactor,                   
                               int numPMs,
                               int lrfSizePerPM,
                               const int nintx,
                               const int ninty,
                               const float* const d_eventsData,
                               int numEvents,
                               float* d_recX,
                               float* d_recY,
                               float* d_recEnergy,
                               float* d_chi2,
                               float* d_probability)

{
  //setting up shared memory
  int activeThreads = blockDim.x * blockDim.y;

  extern __shared__ float shared[];  //these data are shared withing a thread block - that is for all points of the grid (one event!)
    //grid location
  int* nodeIX = (int*) &shared;
  int* nodeIY = (int*) &nodeIX[activeThreads];
  float* nodeChi2 = (float*) &nodeIY[activeThreads];
  float* nodeEnergy = (float*) &nodeChi2[activeThreads];
    //PM
  float* signal = (float*) &nodeEnergy[activeThreads];
    //used only if ML: probability
  float* nodeProb = (float*) &signal[numPMs]; //will be non-zero only for ML!

  __shared__ float Xoffset;
  __shared__ float Yoffset;

  int ievent = blockIdx.x + gridDim.x * blockIdx.y;      //cuda GRID is used to select an event - below word "grid" refers to the grid of XYs positions
  int threadID = threadIdx.x + threadIdx.y * blockDim.x; //block is used to scan XY (each node - one thread)
                                                         //threadIdx.x is related to offset in X from Xcenter of the grid
                                                         //threadIdx.y is related to offset in Y from Ycenter of the grid

  //to do!!! case when numPMs>numtreads in block
  if (threadID<numPMs)
    { //in this block _only_: threadID is PMs index
      signal[threadID] = d_eventsData[ievent*(numPMs+2) + threadID]; //buffer: signals of all active PMs +XY offset
    }

  if (threadID == 0)
    { //0th tread sets the center of the grid
      Xoffset = d_eventsData[ievent*(numPMs+2) + numPMs];  //offsetX;
      Yoffset = d_eventsData[ievent*(numPMs+2) + numPMs+1];//offsetX;
    }

  __syncthreads(); //need to synchronize - all shared input data are ready

 // float tmp1=-1, tmp2=-1; //test var
  int activePMs;

  //starting multigrid iterations
  for (int iter=0; iter<iterations; iter++)
   {
     activePMs = 0;
     float X = Xoffset + scale * (-0.5*(blockDim.x - 1)+ threadIdx.x); //coordinates of this node (Thread)
     float Y = Yoffset + scale * (-0.5*(blockDim.y - 1)+ threadIdx.y);

     if (ievent<numEvents)
       {
         //since we cannot save lrfs in a fast tmp storage, collecting all statistics in parralel
         //energy: SUM sig(i) / SUM LRF(i)
         float sumsig = 0;
         float sumLRF = 0;
         //probability for ML: SUM sig(i)*ln{LRF(i)*E} - LRF(i)*E = SUM   sig(i)*ln(LRF(i)) + sig(i)*ln(E) - LRF(i)*E
         float sumsigLnLRF = 0;
         //chi2:  SUM   {sig(i) - LRF(i)*E)^2 / LRF(i)E  =
         //       SUM   sig2/lrf/E -2*sig +lrf*E
         float sumsig2overLRF = 0;

         bool isBadNode = false; //can be set to true if range to one of PMs larger than the defininition range of its LRF; or energy < 0
         for (int ipm=0; ipm<numPMs; ipm++)
           {
             float xl = d_LRF[lrfSizePerPM*ipm + lrfSizePerPM-4];
             float xr = d_LRF[lrfSizePerPM*ipm + lrfSizePerPM-3];
             float yl = d_LRF[lrfSizePerPM*ipm + lrfSizePerPM-2];
             float yr = d_LRF[lrfSizePerPM*ipm + lrfSizePerPM-1];

             if (X<xl || X>xr)
               {
                 //outside! cannot use this thread (node) at all
                 isBadNode = true;

                 break;
               }
             if (Y<yl || Y>yr)
               {
                 //outside! cannot use this thread (node) at all
                 isBadNode = true;
                 break;
               }

             float dx = xr - xl;
             float dy = yr - yl;

             float tsig = signal[ipm];
             if (ignoreLowSignalPMs)
                if (tsig < ignoreThreshold)
                  {
                    //signal < threshold, ignoring this PM
                    continue;
                  }
             activePMs++; //this one is active

             sumsig += tsig;

             //LRF calculation
             float xi = (X-xl)/dx*nintx;
             int ix = (int)xi;
             float xf = xi-ix;
             float xff = 1.0 - xf;

             float yi = (Y-yl)/dy*ninty;
             int iy = (int)yi;
             float yf = yi-iy;
             float yff = 1.0 - yf;

             float xx[4], yy[4];

             xx[0] = xff*xff*xff;
             xx[1] = xf*xf*(xf + xf + xf - 6.0) + 4.0;
             xx[2] = xf*(xf*(-xf - xf - xf + 3.0) + 3.0) + 1.;
             xx[3] = xf*xf*xf;

             yy[0] = yff*yff*yff;
             yy[1] = yf*yf*(yf + yf + yf - 6.0) + 4.;
             yy[2] = yf*(yf*(-yf - yf - yf + 3.0) + 3.0) + 1.0;
             yy[3] = yf*yf*yf;

             float lrf = 0.0;
             int nbasx = nintx + 3;
             int k = iy*nbasx+ix; // current 2D base function
             for (int jy=0; jy<4; jy++)
               {
                  for (int jx=0; jx<4; jx++)
                    {
                      lrf += d_LRF[lrfSizePerPM*ipm + k] * xx[jx] * yy[jy];
                      k++;
                    }
                  k += nbasx  -4;
               }
             lrf /= 36.0;

       //      if (threadID == 0 && ipm == 0) tmp1 = k;//lrf;
       //      if (threadID == 0 && ipm == 1) tmp2 = lrf;//lrf;

             //----have lrf now----
             if (lrf <= 0)
               { //bad lrf
                 isBadNode = true;
                 break;
               }

             sumLRF += lrf;
             if (mlORchi2 == 0) sumsigLnLRF += tsig * __logf(lrf);  //ML
             sumsig2overLRF += tsig * tsig / lrf;                   //LS

           } //end cycle by PMs

         //can calculate energy ("naive" approach)
         float energy = 1.0;
         if (sumLRF > 0 ) energy = sumsig/sumLRF;
         if (energy<1.0e-10) isBadNode = true;

         //can calculate probability and chi2
         //storing results in shared memory
         if (isBadNode)
           {
             if (mlORchi2 == 0) nodeProb[threadID] = -1.0e10;
             nodeChi2[threadID] = 1.0e10;
           }
         else
           {
             if (mlORchi2 == 0) nodeProb[threadID] = sumsigLnLRF  +  sumsig * __logf(energy)  -  sumLRF*energy;
             nodeChi2[threadID] = sumsig2overLRF/energy -2*sumsig + sumLRF*energy;
           }

         nodeIX[threadID] = threadIdx.x;
         nodeIY[threadID] = threadIdx.y;
         nodeEnergy[threadID] = energy;         
       }
     else
       {
         //could happen for num events >65535
         //wrong event index - we dont care, these data are not sent to host
         //only important: these threads still can see all __syncthreads()
       }

     __syncthreads(); //to prepare node data

     //looking for the best location - compare nodes always in pairs - using shared memory
      int Nodes = activeThreads;    //activeThreads = blockDim.x * blockDim.y
      do
        {
          int idx = Nodes / 2;
          //---

          if (mlORchi2 == 0) //ML
            {
              if (threadID < idx && nodeProb[threadID] < nodeProb[threadID + idx])
                {
                 nodeProb[threadID] = nodeProb[threadID + idx];
                 nodeIX[threadID] = nodeIX[threadID + idx];
                 nodeIY[threadID] = nodeIY[threadID + idx];
                 nodeEnergy[threadID] = nodeEnergy[threadID + idx];
                 nodeChi2[threadID] = nodeChi2[threadID + idx];
                }

              if (threadID == 0)
               if (Nodes % 2 != 0)  //for the cases when Nodes is odd number
                {
                  if (nodeProb[0] < nodeProb[Nodes-1]) //all other nodes are already covered
                    {
                     nodeProb[0] = nodeProb[Nodes-1];
                     nodeIX[0] = nodeIX[Nodes-1];
                     nodeIY[0] = nodeIY[Nodes-1];
                     nodeEnergy[0] = nodeEnergy[Nodes-1];
                     nodeChi2[0] = nodeChi2[Nodes-1];
                    }
                }
            }
          else
            {
              if (threadID < idx && nodeChi2[threadID] > nodeChi2[threadID + idx])
                {
                 nodeIX[threadID] = nodeIX[threadID + idx];
                 nodeIY[threadID] = nodeIY[threadID + idx];
                 nodeEnergy[threadID] = nodeEnergy[threadID + idx];
                 nodeChi2[threadID] = nodeChi2[threadID + idx];
                }

              if (threadID == 0)
               if (Nodes % 2 != 0)  //for the cases when Nodes is odd number
                {
                  if (nodeChi2[0] > nodeChi2[Nodes-1]) //all other nodes are already covered
                    {
                     nodeIX[0] = nodeIX[Nodes-1];
                     nodeIY[0] = nodeIY[Nodes-1];
                     nodeEnergy[0] = nodeEnergy[Nodes-1];
                     nodeChi2[0] = nodeChi2[Nodes-1];
                    }
                }
            }


          //---
          Nodes = idx;
          __syncthreads();
        }
      while (Nodes>1);

     // thread index 0 contains info on to the best fit location with the current grid
     if (threadID == 0)
       {
         Xoffset +=  scale * (-0.5*(blockDim.x - 1)+ nodeIX[0]); //new gid center - for the next iteration
         Yoffset +=  scale * (-0.5*(blockDim.y - 1)+ nodeIY[0]);
       }
     __syncthreads();


     //preparing scale for the next iteration
     scale /= scaleReductionFactor;
   }//iteration end

  //reporting back reconstructed event data
  if (threadID == 0 && ievent<numEvents)
    {
      //reporting results
      d_recX[ievent] = Xoffset;
      d_recY[ievent] = Yoffset;
      d_recEnergy[ievent] = nodeEnergy[0];
      int df = activePMs - 4; //-1 -2XY -1energy
      if (df < 1) df = 1;
      d_chi2[ievent] = nodeChi2[0] / df;
      if (mlORchi2 == 0) d_probability[ievent] = nodeProb[0];
    }
}
*/


//same as Freeform, but LRFs are given in local coordinates of the sensors
__global__ void kernelXY(const bool mlORchi2,
                               const bool ignoreLowSignalPMs,
                               const float ignoreThresholdLow,
                               const float ignoreThresholdHigh,
                         const bool ignoreFarPMs,
                         const float ignoreDistance2,
                         const float* pmx,
                         const float* pmy,
                               const int iterations,
                               float scale,
                               const float scaleReductionFactor,
                               int numPMs,
                               int lrfSizePerPM,
                               const int nintx,
                               const int ninty,
                               const float* const d_lrfData,
                               const float* const d_eventsData,
                               int numEvents,
                               float* d_recX,
                               float* d_recY,
                               float* d_recEnergy,
                               float* d_chi2,
                               float* d_probability)

{
  //setting up shared memory
  int activeThreads = blockDim.x * blockDim.y;    
  extern __shared__ float shared[];  //these data are shared withing a thread block - that is for all points of the grid (one event!)
    //PM centers
 float* PMx = (float*) &shared;
 float* PMy = (float*) &PMx[numPMs];
    //grid location
  //int* nodeIX = (int*) &shared;
  int* nodeIX = (int*) &PMy[numPMs];
  int* nodeIY = (int*) &nodeIX[activeThreads];
  float* nodeChi2 = (float*) &nodeIY[activeThreads];
  float* nodeEnergy = (float*) &nodeChi2[activeThreads];
    //PM
  float* signal = (float*) &nodeEnergy[activeThreads];
    //used only if ML: probability
  float* nodeProb = (float*) &signal[numPMs]; //will be non-zero only for ML!

  __shared__ float Xoffset;
  __shared__ float Yoffset;
  __shared__ float Xoffset0; //will be kept unchanged, for dynamic passives by distance
  __shared__ float Yoffset0; //will be kept unchanged, for dynamic passives by distance

  int ievent = blockIdx.x + gridDim.x * blockIdx.y;      //cuda GRID is used to select an event - below word "grid" refers to the grid of XYs positions
  int threadID = threadIdx.x + threadIdx.y * blockDim.x; //block is used to scan XY (each node - one thread)
                                                         //threadIdx.x is related to offset in X from Xcenter of the grid
                                                         //threadIdx.y is related to offset in Y from Ycenter of the grid

  //to do!!! case when numPMs>numtreads in block
  if (threadID<numPMs)
    { //in this block _only_: threadID is PMs index
      signal[threadID] = d_eventsData[ievent*(numPMs+2) + threadID]; //buffer: signals of all active PMs +XY offset
    }

  if (threadID == 0)
    { //0th tread sets the center of the grid
      Xoffset = d_eventsData[ievent*(numPMs+2) + numPMs];  //offsetX;
      Xoffset0 = Xoffset;
      Yoffset = d_eventsData[ievent*(numPMs+2) + numPMs+1];//offsetY;
      Yoffset0 = Yoffset;
    }

  __syncthreads(); //need to synchronize - all shared input data are ready

 // float tmp1=-1, tmp2=-1; //test var
  int activePMs;

  //starting multigrid iterations
  for (int iter=0; iter<iterations; iter++)
   {
     activePMs = 0;
     float X = Xoffset + scale * (-0.5*(blockDim.x - 1)+ threadIdx.x); //coordinates of this node (Thread)
     float Y = Yoffset + scale * (-0.5*(blockDim.y - 1)+ threadIdx.y);

     if (ievent<numEvents)
       {
         //since we cannot save lrfs in a fast tmp storage, collecting all statistics in parralel
         //energy: SUM sig(i) / SUM LRF(i)
         float sumsig = 0;
         float sumLRF = 0;
         //probability for ML: SUM sig(i)*ln{LRF(i)*E} - LRF(i)*E = SUM   sig(i)*ln(LRF(i)) + sig(i)*ln(E) - LRF(i)*E
         float sumsigLnLRF = 0;
         //chi2:  SUM   {sig(i) - LRF(i)*E)^2 / LRF(i)E  =
         //       SUM   sig2/lrf/E -2*sig +lrf*E
         float sumsig2overLRF = 0;

         bool isBadNode = false; //can be set to true if range to one of PMs larger than the defininition range of its LRF; or energy < 0
         for (int ipm=0; ipm<numPMs; ipm++)
           {
             //doing backtransform
             float deltax = d_lrfData[lrfSizePerPM*ipm + lrfSizePerPM-9];
             float deltay = d_lrfData[lrfSizePerPM*ipm + lrfSizePerPM-8];
             float sinphi = d_lrfData[lrfSizePerPM*ipm + lrfSizePerPM-7];
             float cosphi = d_lrfData[lrfSizePerPM*ipm + lrfSizePerPM-6];
             float flip =   d_lrfData[lrfSizePerPM*ipm + lrfSizePerPM-5];

             float xtmp = X + deltax;
             float ytmp = Y + deltay;
             float       XL =  xtmp*cosphi - ytmp*sinphi;
             float YL;
             if (flip<0) YL =  xtmp*sinphi + ytmp*cosphi;  //no flip
             else        YL = -xtmp*sinphi - ytmp*cosphi;  //with flip

             float xl = d_lrfData[lrfSizePerPM*ipm + lrfSizePerPM-4];
             float xr = d_lrfData[lrfSizePerPM*ipm + lrfSizePerPM-3];
             float yl = d_lrfData[lrfSizePerPM*ipm + lrfSizePerPM-2];
             float yr = d_lrfData[lrfSizePerPM*ipm + lrfSizePerPM-1];

             if (XL<xl || XL>xr)
               {
                 //outside! cannot use this thread (node) at all
                 isBadNode = true;

                 break;
               }
             if (YL<yl || YL>yr)
               {
                 //outside! cannot use this thread (node) at all
                 isBadNode = true;
                 break;
               }

             float dx = xr - xl;
             float dy = yr - yl;

             float tsig = signal[ipm];
             if (ignoreLowSignalPMs)
                if (tsig < ignoreThresholdLow || tsig > ignoreThresholdHigh)
                  {
                    //signal < threshold, ignoring this PM
                    continue;
                  }
             if (ignoreFarPMs)
               {
                 float stX  = (PMx[ipm] - Xoffset0);
                 float stY  = (PMy[ipm] - Yoffset0);
                 float r2 = stX*stX + stY*stY; // r  = sqrtf(r);
                 //distance2 from this PM center to the start position of the search
                 if (r2 > ignoreDistance2)
                  {
                   //too far from this PM center, ignoring this PM
                   continue;
                  }
               }

             activePMs++; //this one is active

             sumsig += tsig;

             //LRF calculation
             float xi = (XL-xl)/dx*nintx;
             int ix = (int)xi;
             float xf = xi-ix;
             float xff = 1.0 - xf;

             float yi = (YL-yl)/dy*ninty;
             int iy = (int)yi;
             float yf = yi-iy;
             float yff = 1.0 - yf;

             float xx[4], yy[4];

             xx[0] = xff*xff*xff;
             xx[1] = xf*xf*(xf + xf + xf - 6.0) + 4.0;
             xx[2] = xf*(xf*(-xf - xf - xf + 3.0) + 3.0) + 1.;
             xx[3] = xf*xf*xf;

             yy[0] = yff*yff*yff;
             yy[1] = yf*yf*(yf + yf + yf - 6.0) + 4.;
             yy[2] = yf*(yf*(-yf - yf - yf + 3.0) + 3.0) + 1.0;
             yy[3] = yf*yf*yf;

             float lrf = 0.0;
             int nbasx = nintx + 3;
             int k = iy*nbasx+ix; // current 2D base function
             for (int jy=0; jy<4; jy++)
               {
                  for (int jx=0; jx<4; jx++)
                    {
                      lrf += d_lrfData[lrfSizePerPM*ipm + k] * xx[jx] * yy[jy];
                      k++;
                    }
                  k += nbasx  -4;
               }
             lrf /= 36.0;

       //      if (threadID == 0 && ipm == 0) tmp1 = k;//lrf;
       //      if (threadID == 0 && ipm == 1) tmp2 = lrf;//lrf;

             //----have lrf now----
             if (lrf <= 0)
               { //bad lrf
                 isBadNode = true;
                 break;
               }

             sumLRF += lrf;
             if (mlORchi2 == 0) sumsigLnLRF += tsig * __logf(lrf);  //ML
             sumsig2overLRF += tsig * tsig / lrf;                   //LS

           } //end cycle by PMs

         //can calculate energy ("naive" approach)
         float energy = 1.0;
         if (sumLRF > 0 ) energy = sumsig/sumLRF;
         if (energy<1.0e-10) isBadNode = true;

         //can calculate probability and chi2
         //storing results in shared memory
         if (isBadNode)
           {
             if (mlORchi2 == 0) nodeProb[threadID] = -1.0e10;
             nodeChi2[threadID] = 1.0e10;
           }
         else
           {
             if (mlORchi2 == 0) nodeProb[threadID] = sumsigLnLRF  +  sumsig * __logf(energy)  -  sumLRF*energy;
             nodeChi2[threadID] = sumsig2overLRF/energy -2*sumsig + sumLRF*energy;
           }

         nodeIX[threadID] = threadIdx.x;
         nodeIY[threadID] = threadIdx.y;
         nodeEnergy[threadID] = energy;
       }
     else
       {
         //could happen for num events >65535
         //wrong event index - we dont care, these data are not sent to host
         //only important: these threads still can see all __syncthreads()
       }

     __syncthreads(); //to prepare node data

     //looking for the best location - compare nodes always in pairs - using shared memory
      int Nodes = activeThreads;    //activeThreads = blockDim.x * blockDim.y
      do
        {
          int idx = Nodes / 2;
          //---

          if (mlORchi2 == 0) //ML
            {
              if (threadID < idx && nodeProb[threadID] < nodeProb[threadID + idx])
                {
                 nodeProb[threadID] = nodeProb[threadID + idx];
                 nodeIX[threadID] = nodeIX[threadID + idx];
                 nodeIY[threadID] = nodeIY[threadID + idx];
                 nodeEnergy[threadID] = nodeEnergy[threadID + idx];
                 nodeChi2[threadID] = nodeChi2[threadID + idx];
                }

              if (threadID == 0)
               if (Nodes % 2 != 0)  //for the cases when Nodes is odd number
                {
                  if (nodeProb[0] < nodeProb[Nodes-1]) //all other nodes are already covered
                    {
                     nodeProb[0] = nodeProb[Nodes-1];
                     nodeIX[0] = nodeIX[Nodes-1];
                     nodeIY[0] = nodeIY[Nodes-1];
                     nodeEnergy[0] = nodeEnergy[Nodes-1];
                     nodeChi2[0] = nodeChi2[Nodes-1];
                    }
                }
            }
          else
            {
              if (threadID < idx && nodeChi2[threadID] > nodeChi2[threadID + idx])
                {
                 nodeIX[threadID] = nodeIX[threadID + idx];
                 nodeIY[threadID] = nodeIY[threadID + idx];
                 nodeEnergy[threadID] = nodeEnergy[threadID + idx];
                 nodeChi2[threadID] = nodeChi2[threadID + idx];
                }

              if (threadID == 0)
               if (Nodes % 2 != 0)  //for the cases when Nodes is odd number
                {
                  if (nodeChi2[0] > nodeChi2[Nodes-1]) //all other nodes are already covered
                    {
                     nodeIX[0] = nodeIX[Nodes-1];
                     nodeIY[0] = nodeIY[Nodes-1];
                     nodeEnergy[0] = nodeEnergy[Nodes-1];
                     nodeChi2[0] = nodeChi2[Nodes-1];
                    }
                }
            }


          //---
          Nodes = idx;
          __syncthreads();
        }
      while (Nodes>1);

     // thread index 0 contains info on to the best fit location with the current grid
     if (threadID == 0)
       {
         Xoffset +=  scale * (-0.5*(blockDim.x - 1)+ nodeIX[0]); //new gid center - for the next iteration
         Yoffset +=  scale * (-0.5*(blockDim.y - 1)+ nodeIY[0]);
       }
     __syncthreads();


     //preparing scale for the next iteration
     scale /= scaleReductionFactor;
   }//iteration end

  //reporting back reconstructed event data
  if (threadID == 0 && ievent<numEvents)
    {
      //reporting results
      d_recX[ievent] = Xoffset;
      d_recY[ievent] = Yoffset;
      d_recEnergy[ievent] = nodeEnergy[0];
      int df = activePMs -4; //-1 -2XY -1energy
      if (df < 1) df = 1;
      d_chi2[ievent] = nodeChi2[0] / df;
      if (mlORchi2 == 0) d_probability[ievent] = nodeProb[0];
    }
}

__global__ void kernelComposite(const bool mlORchi2,
                                const bool ignoreLowSigPMs,
                                const float ignoreThresholdLow,
                                const float ignoreThresholdHigh,
                                const bool ignoreFarPMs,
                                const float ignoreDistance2,
                                const int iterations,
                                float scale,
                                const float scaleReductionFactor,
                                const float comp_r0,
                                const float comp_a,
                                const float comp_b,
                                const float comp_lam2,
                                const bool fCompressed,
                                const float* pmx,
                                const float* pmy,
                                int numPMs,
                                int lrfFloatsPerPM,
                                int lrfFloatsAxialPerPM,
                                int nintx,
                                int ninty,
                                const float* const d_lrfData,
                                const float* const d_eventsData,
                                int numEvents,
                                float* d_recX,
                                float* d_recY,
                                float* d_recEnergy,
                                float* d_chi2,
                                float* d_probability)
{
  //setting up shared memory
  int activeThreads = blockDim.x * blockDim.y;

  extern __shared__ float shared[];  //these data are shared withing a thread block - that is for all points of the grid (one event!)
  float* PMx = (float*) &shared;
  float* PMy = (float*) &PMx[numPMs];
  float* signal = (float*) &PMy[numPMs];
  int* nodeIX = (int*) &signal[numPMs];
  int* nodeIY = (int*) &nodeIX[activeThreads];
  float* nodeChi2 = (float*) &nodeIY[activeThreads];
  float* nodeEnergy = (float*) &nodeChi2[activeThreads];
    //for ML only
  float* nodeProb = (float*) &nodeEnergy[activeThreads];

  __shared__ float Xoffset;
  __shared__ float Yoffset;
  __shared__ float Xoffset0; //will be kept unchanged, for dynamic passives by distance
  __shared__ float Yoffset0; //will be kept unchanged, for dynamic passives by distance

  int ievent = blockIdx.x + gridDim.x * blockIdx.y;      //cuda GRID is used to select an event - below word "grid" refers to the grid of XYs positions
  int threadID = threadIdx.x + threadIdx.y * blockDim.x; //block is used to scan XY (each node - one thread)
                                                         //threadIdx.x is related to offset in X from Xcenter of the grid
                                                         //threadIdx.y is related to offset in Y from Ycenter of the grid

  //to do!!! case when numPMs>numtreads in block
  if (threadID<numPMs)
    { //in this block _only_: threadID is PMs index
      PMx[threadID] = pmx[threadID];
      PMy[threadID] = pmy[threadID];

      signal[threadID] = d_eventsData[ievent*(numPMs+2) + threadID]; //buffer contains PM signals and XY offset
    }

  if (threadID == 0)
    { //0th tread sets the center of the grid
      Xoffset = d_eventsData[ievent*(numPMs+2) + numPMs];  //offsetX;
      Xoffset0 = Xoffset;
      Yoffset = d_eventsData[ievent*(numPMs+2) + numPMs+1];//offsetY;
      Yoffset0 = Yoffset;
    }

  __syncthreads(); //need to synchronize - all shared input data are ready

  int activePMs;

  //starting multigrid iterations
  for (int iter=0; iter<iterations; iter++)
   {
     float X = Xoffset + scale * (-0.5*(blockDim.x - 1)+ threadIdx.x); //coordinates of this node (Thread)
     float Y = Yoffset + scale * (-0.5*(blockDim.y - 1)+ threadIdx.y);

     if (ievent<numEvents)
       {
         activePMs = 0;
         //since we cannot save lrfs in a fast tmp storage, collecting all statistics in parralel
         //energy - SUM sig(i) / SUM LRF(i)
         float sumsig = 0;
         float sumLRF = 0;
         //probability for ML: SUM sig(i)*ln{LRF(i)*E} - LRF(i)*E = SUM   sig(i)*ln(LRF(i)) + sig(i)*ln(E) - LRF(i)*E
         float sumsigLnLRF = 0;
         //for chi2:  SUM   {sig(i) - LRF(i)*E)^2 / LRF(i)E  =
         //           SUM   sig2/lrf/E -2*sig +lrf*E
         float sumsig2overLRF = 0;

         bool isBadNode = false; //can be set to true if range to one of PMs larger than the defininition range of its LRF; or energy < 0
         for (int ipm=0; ipm<numPMs; ipm++)
           {
             float tsig = signal[ipm];

             //check if this PM is skipped..
             if (ignoreLowSigPMs)       //..by signal
                if (tsig < ignoreThresholdLow || tsig > ignoreThresholdHigh)
                 {
                  //signal < threshold, ignoring this PM
                  continue;
                 }
             if (ignoreFarPMs)          //..by distance
               {
                 float stX  = (PMx[ipm] - Xoffset0);
                 float stY  = (PMy[ipm] - Yoffset0);
                 float r2 = stX*stX + stY*stY; // r  = sqrtf(r);
                 //distance2 from this PM center to the start position of the search
                 if (r2 > ignoreDistance2)
                  {
                   //too far from this PM center, ignoring this PM
                   continue;
                  }
               }

             //check if both LRF are defined and applying conversions
             //..first axial
             float dr = d_lrfData[lrfFloatsPerPM*ipm + lrfFloatsAxialPerPM-1]; //LRFs are defined in the range from 0 to dr
             float offX  = (X - PMx[ipm]);
             float offY  = (Y - PMy[ipm]);
             float r = offX*offX + offY*offY;
             r  = sqrtf(r); //distance to the PM center
             if (r > dr)
               {
                 //outside! cannot use this thread (node) at all
                 isBadNode = true;
                 break;
               }
               //applying compression
             if (fCompressed)
               {
                 float delta_r = r - comp_r0;
                 r =  comp_b + delta_r*comp_a - sqrtf(delta_r*delta_r + comp_lam2);
                 if (r<0.0) r = 0.0;

                 delta_r = dr - comp_r0;
                 dr =  comp_b + delta_r*comp_a - sqrtf(delta_r*delta_r + comp_lam2);
                 // if (dr<0.0) dr = 0.0;
               }
             //..then xy
               //doing backtransform
             float deltax = d_lrfData[lrfFloatsPerPM*ipm + lrfFloatsPerPM-9];
             float deltay = d_lrfData[lrfFloatsPerPM*ipm + lrfFloatsPerPM-8];
             float sinphi = d_lrfData[lrfFloatsPerPM*ipm + lrfFloatsPerPM-7];
             float cosphi = d_lrfData[lrfFloatsPerPM*ipm + lrfFloatsPerPM-6];
             float flip =   d_lrfData[lrfFloatsPerPM*ipm + lrfFloatsPerPM-5];

             float xtmp = X + deltax;
             float ytmp = Y + deltay;
             float       XL =  xtmp*cosphi - ytmp*sinphi;
             float YL;
             if (flip<0) YL =  xtmp*sinphi + ytmp*cosphi;  //no flip
             else        YL = -xtmp*sinphi - ytmp*cosphi;  //with flip

             float xl = d_lrfData[lrfFloatsPerPM*ipm + lrfFloatsPerPM-4];
             float xr = d_lrfData[lrfFloatsPerPM*ipm + lrfFloatsPerPM-3];
             float yl = d_lrfData[lrfFloatsPerPM*ipm + lrfFloatsPerPM-2];
             float yr = d_lrfData[lrfFloatsPerPM*ipm + lrfFloatsPerPM-1];

             //if (threadID == 0) printf("PM#: %d \n XY:%f, %f\n FXL,YL: %f %f\nRange minmax X: %f %f\nY: %f %f\n",ipm, X, Y, XL, YL, xl,xr,yl,yr);

             if (XL<xl || XL>xr)
               { //outside! cannot use this thread (node) at all
                 isBadNode = true;
                 break;
               }
             if (YL<yl || YL>yr)
               { //outside! cannot use this thread (node) at all
                 isBadNode = true;
                 break;
               }

             //if we reach this point, this PM is active!
             activePMs++;
             sumsig += tsig;


             //LRF calculation..
             //..axial part
             if (r == dr) dr -= 1.0e-7;
             int nint = lrfFloatsAxialPerPM-4; // lrfFloatsAxialPerPM = number of intervals +3 + 1
             float xi = r/dr*nint;
             int ix = (int)xi;
             float xf = xi-ix;

             float a0, a1, a2;
             float p0, p1, p2, p3;

             a0 = d_lrfData[lrfFloatsPerPM*ipm + ix];
             a1 = d_lrfData[lrfFloatsPerPM*ipm + ix + 1];
             float c2 = d_lrfData[lrfFloatsPerPM*ipm + ix + 2];
             a2 = a0 + a1 + c2;  // c0 + c1 + c2
             a1 = a1 + a1 + a1;  // 3*c1
             p0 = a2 + a1;       // c0 + 4*c1 + c2
             p2 = a2 - a1;       // c0 - 2*c1 + c2
             p2 = p2 + p2 + p2;  // 3*c0 - 6*c1 + 3*c2
             p1 = c2 - a0;       // c2 - c0
             p1 = p1 + p1 + p1;  // 3*c2 - 3*c0
             p3 = a0 + a0 - a1 - p2 + d_lrfData[lrfFloatsPerPM*ipm + ix + 3]; // -c0 + 3*c1 - 3*c2 + c3

             float lrfAxial = (p0 + xf*(p1 + xf*(p2 + xf*p3)))/6.;
//             if (lrfAxial <= 0)
//               { //bad lrf
//                 isBadNode = true;
//                 break;
//               }

             //..now xy LRFs
             float dx = xr - xl;
             float dy = yr - yl;
             xi = (XL-xl)/dx*nintx;
             ix = (int)xi;
             xf = xi-ix;
             float xff = 1.0 - xf;

             float yi = (YL-yl)/dy*ninty;
             int iy = (int)yi;
             float yf = yi-iy;
             float yff = 1.0 - yf;

             float xx[4], yy[4];

             xx[0] = xff*xff*xff;
             xx[1] = xf*xf*(xf + xf + xf - 6.0) + 4.0;
             xx[2] = xf*(xf*(-xf - xf - xf + 3.0) + 3.0) + 1.;
             xx[3] = xf*xf*xf;

             yy[0] = yff*yff*yff;
             yy[1] = yf*yf*(yf + yf + yf - 6.0) + 4.;
             yy[2] = yf*(yf*(-yf - yf - yf + 3.0) + 3.0) + 1.0;
             yy[3] = yf*yf*yf;

             float lrfXY = 0.0;
             int nbasx = nintx + 3;
             int k = iy*nbasx+ix; // current 2D base function
             for (int jy=0; jy<4; jy++)
               {
                  for (int jx=0; jx<4; jx++)
                    {
                      lrfXY += d_lrfData[lrfFloatsPerPM*ipm + lrfFloatsAxialPerPM + k] * xx[jx] * yy[jy];
                      k++;
                    }
                  k += nbasx  -4;
               }
             lrfXY /= 36.0;

             float lrf = lrfAxial + lrfXY;
             //note - it seems now as it is implemented in the LRF library, individual LRFs can be negative, only important that sum LRFs is > 0
             if (lrf <= 0)
               { //bad lrf
                 isBadNode = true;
                 break;
               }


             //--------------- have lrf now ----------------------
             //printf("Pm number: %d, XY: %f,%f, LRF=%f\n", ipm, X, Y, lrf);

             sumLRF += lrf;
             if (mlORchi2 == 0) sumsigLnLRF += tsig * __logf(lrf);
             sumsig2overLRF += tsig * tsig / lrf;

           } //end cycle by PMs

         //can calculate energy ("naive" approach)
         float energy = 1.0;
         if (sumLRF > 0 ) energy = sumsig/sumLRF;
         if (energy<1.0e-10) isBadNode = true;

         //can calculate probability and chi2
         //storing results in shared memory
         if (isBadNode)
           {
             if (mlORchi2 == 0) nodeProb[threadID] = -1.0e10;
             nodeChi2[threadID] = 1.0e10;
           }
         else
           {
             if (mlORchi2 == 0) nodeProb[threadID] = sumsigLnLRF  +  sumsig * __logf(energy)  -  sumLRF*energy;
             nodeChi2[threadID] = sumsig2overLRF/energy -2*sumsig + sumLRF*energy;
           }

         nodeIX[threadID] = threadIdx.x;
         nodeIY[threadID] = threadIdx.y;
         nodeEnergy[threadID] = energy;
       }
     else
       {
         //could happen for num events >65535
         //wrong event index - we dont care, these data are not sent to host
         //only important: these threads still can see all __syncthreads()
       }

     __syncthreads(); //to prepare node data

     //looking for the best location - compare nodes always in pairs - using shared memory
      int Nodes = activeThreads;    //activeThreads = blockDim.x * blockDim.y
      do
              {
                int idx = Nodes / 2;
                //---

                if (mlORchi2 == 0) //ML
                  {
                    if (threadID < idx && nodeProb[threadID] < nodeProb[threadID + idx])
                      {
                       nodeProb[threadID] = nodeProb[threadID + idx];
                       nodeIX[threadID] = nodeIX[threadID + idx];
                       nodeIY[threadID] = nodeIY[threadID + idx];
                       nodeEnergy[threadID] = nodeEnergy[threadID + idx];
                       nodeChi2[threadID] = nodeChi2[threadID + idx];
                      }

                    if (threadID == 0)
                     if (Nodes % 2 != 0)  //for the cases when Nodes is odd number
                      {
                        if (nodeProb[0] < nodeProb[Nodes-1]) //all other nodes are already covered
                          {
                           nodeProb[0] = nodeProb[Nodes-1];
                           nodeIX[0] = nodeIX[Nodes-1];
                           nodeIY[0] = nodeIY[Nodes-1];
                           nodeEnergy[0] = nodeEnergy[Nodes-1];
                           nodeChi2[0] = nodeChi2[Nodes-1];
                          }
                      }
                  }
                else
                  {
                    if (threadID < idx && nodeChi2[threadID] > nodeChi2[threadID + idx])
                      {
                       nodeIX[threadID] = nodeIX[threadID + idx];
                       nodeIY[threadID] = nodeIY[threadID + idx];
                       nodeEnergy[threadID] = nodeEnergy[threadID + idx];
                       nodeChi2[threadID] = nodeChi2[threadID + idx];
                      }

                    if (threadID == 0)
                     if (Nodes % 2 != 0)  //for the cases when Nodes is odd number
                      {
                        if (nodeChi2[0] > nodeChi2[Nodes-1]) //all other nodes are already covered
                          {
                           nodeIX[0] = nodeIX[Nodes-1];
                           nodeIY[0] = nodeIY[Nodes-1];
                           nodeEnergy[0] = nodeEnergy[Nodes-1];
                           nodeChi2[0] = nodeChi2[Nodes-1];
                          }
                      }
                  }


                //---
                Nodes = idx;
                __syncthreads();
              }
            while (Nodes>1);

     // thread index 0 contains info on to the best fit location with the current grid
     if (threadID == 0)
       {
         Xoffset +=  scale * (-0.5*(blockDim.x - 1)+ nodeIX[0]); //new gid center - for the next iteration
         Yoffset +=  scale * (-0.5*(blockDim.y - 1)+ nodeIY[0]);
       }
     __syncthreads();


     //preparing scale for the next iteration
     scale /= scaleReductionFactor;
   }//iteration end

  //reporting back reconstructed event data
  if (threadID == 0 && ievent<numEvents)
      {
        //reporting results
        d_recX[ievent] = Xoffset;
        d_recY[ievent] = Yoffset;
        d_recEnergy[ievent] = nodeEnergy[0];
        int df = activePMs -4;  //-1 -2XY -1energy
        if (df < 1) df = 1;
        d_chi2[ievent] = nodeChi2[0] / df;
        if (mlORchi2 == 0) d_probability[ievent] = nodeProb[0];

        //printf("XY: %f,%f, Energy:%f\nChi2:%f\nActive PMs:%d\n", Xoffset, Yoffset, nodeEnergy[0], nodeChi2[0]/df, activePMs);
      }
}
